#include "iteration_host.h"
#include "iteration_kernel.cuh"
#include "lbm_model.cuh"

const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
const dim3 gridSize(((lx + BLKXSIZE - 1) / BLKXSIZE),
                    ((ly + BLKYSIZE - 1) / BLKYSIZE),
                    ((lz + BLKZSIZE - 1) / BLKZSIZE));

//! Invoke double grids to excute the streaming
const dim3 gridSize_Q(((SCALE_LX * lx + BLKXSIZE - 1) / BLKXSIZE),
                      ((ly + BLKYSIZE - 1) / BLKYSIZE),
                      ((lz + BLKZSIZE - 1) / BLKZSIZE));

//! Copy constant parameters to global symbols
void transfer_parameter(real* tau, real* G, bool* carn_star, real* T_Tc,
                        real* rhol_spinodal, real* rhog_spinodal,
                        bool* inject_density, size_t* rho_inject_period,
                        real* rho_increment, real* rhos, real* inject_position,
                        size_t* statPeriod) {

  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dtau), tau, sizeof(real)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dG), G, sizeof(real)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dcarn_star), carn_star, sizeof(bool)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dT_Tc), T_Tc, sizeof(real)));
  cudaErrorCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(drhol_spinodal), rhol_spinodal, sizeof(real)));
  cudaErrorCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(drhog_spinodal), rhog_spinodal, sizeof(real)));
  cudaErrorCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(dinject_density), inject_density, sizeof(bool)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(drho_inject_period), rho_inject_period,
                                    sizeof(size_t)));
  cudaErrorCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(drho_increment), rho_increment, sizeof(real)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(drhos), rhos, sizeof(real)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dinject_position), inject_position,
                                    sizeof(unsigned int)));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dstatPeriod), statPeriod, sizeof(size_t)));
  hipDeviceSynchronize();
}

//! Initialize multidimensional arrays on GPU
void multiphaselbm_initialization(Realxyz** rho_dd, Realxyz** psi_dd,
                                  Realxyz** pressure_dd, Boolxyz** is_fluid_dd,

                                  RealxyzQ** collision_f_dd,
                                  RealxyzQ** stream_f_dd, Realxyz3** force_dd) {

  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(rho_d), rho_dd, sizeof(*rho_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(psi_d), psi_dd, sizeof(*psi_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(pressure_d), pressure_dd,
                                    sizeof(*pressure_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(is_fluid_d), is_fluid_dd,
                                    sizeof(*is_fluid_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(collision_f_d), collision_f_dd,
                                    sizeof(*collision_f_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(stream_f_d), stream_f_dd,
                                    sizeof(*stream_f_dd), 0,
                                    hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(force_d), force_dd, sizeof(*force_dd), 0,
                                    hipMemcpyHostToDevice));

  multiphaselbm_init<<<gridSize, blockSize, 0>>>();
}

//! Invoke kernel functions to run iterations on GPU
void multiphaselbm_iteration(long step, size_t statPeriod, int reverse,
                             unsigned int inject_type) {
  // The inject type can be changed under given condition
  dinject_type = inject_type;
  do_streaming<<<gridSize_Q, blockSize, 0>>>();
  treat_boundary<<<gridSize, blockSize, 0>>>();
  do_collision<<<gridSize, blockSize, 0>>>(step, reverse);
  do_swap<<<1, 1, 0>>>();
}
